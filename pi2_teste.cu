#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>

// const defines
#define NBIN 1000000000
#define NUM_BLOCK 4
#define NUM_THREAD 16

// struct to get time
struct timeval current_time = {0,0};

int tid;
float pi = 0, time_elapsed;
uint begin, end;

// function to calculate pi
__global__ void cal_pi(double *sum, int nbin, double step, int nthreads, int nblocks) 
{
	// var declaration
	int i;
	double x;
	int idx = blockIdx.x*blockDim.x+threadIdx.x; 
	
	// pi equation calculation
	for (i=idx; i< nbin; i+=nthreads*nblocks) {
		x = i*step;
		sum[idx] += double(4.0/(1.0+(x*x)));
	}
}

int main() 
{
	// var declaration
	double *sumDev;
	double step = 1.0/NBIN; // dx
	size_t size = NUM_BLOCK*NUM_THREAD*sizeof(float);

	// initializing file in append mode to insert experiment data
	FILE *f = fopen("/home/aac-pc/Daniel/experimental-log.txt", "a");
	
	// alooc space to acc variable
	hipMallocManaged(&sumDev, size); 
	
	// get initial time to evaluate performance
	gettimeofday(&current_time, NULL);	
	begin = current_time.tv_sec*1000000 + current_time.tv_usec;
	
	// call function to calculate pi in threads
	cal_pi<<<NUM_BLOCK, NUM_THREAD>>>(sumDev, NBIN, step, NUM_THREAD, NUM_BLOCK);
	// synchronize threads
	hipDeviceSynchronize();
	
	// get final time to evaluate performance
	gettimeofday(&current_time, NULL);
	end = current_time.tv_sec*1000000 + current_time.tv_usec;
	time_elapsed = end - begin;
	
	// calculate pi final value
	for(tid=0; tid<NUM_THREAD*NUM_BLOCK; tid++){
		pi += sumDev[tid];
	}
	pi *= step;
	
	// print final value in console and save data info in log file
	printf("PI = %f\n",pi);
	fprintf(f, "%d;%d;%f;%f\n", NUM_THREAD, NUM_BLOCK, pi, (time_elapsed/1000000));	

	// free cuda var
	hipFree(sumDev);

	// close file
	fclose(f);

	return 0;
}